
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// Kernel function for matrix-vector multiplication using row-wise decomposition
__global__ void matrixVectorMultiply(float *output, float *inputMatrix, float *inputVector, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N) {
        float sum = 0.0f;
        for (int col = 0; col < N; col++) {
            sum += inputMatrix[row * N + col] * inputVector[col];
        }
        output[row] = sum;
    }
}

// Function to generate random values for matrix elements
void generateRandomMatrix(float *matrix, int N) {
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            matrix[i * N + j] = (float)rand() / RAND_MAX; // Random float between 0 and 1
        }
    }
}

int main() {
    int N = 640;

    for (int i = 1; i <= 20; i++) {
        int threadsnum = 32 * i;

        // Start measuring sequential execution time
        clock_t sequential_start = clock();

        // Allocate memory for input matrix, input vector, and output vector
        float *inputMatrix = (float *)malloc(N * N * sizeof(float));
        float *inputVector = (float *)malloc(N * sizeof(float));
        float *outputVector = (float *)malloc(N * sizeof(float));

        // Generate random values for input matrix and input vector
        generateRandomMatrix(inputMatrix, N);
        for (int j = 0; j < N; j++) {
            inputVector[j] = (float)rand() / RAND_MAX; // Random float
        }

        // Allocate memory on the device
        float *d_inputMatrix, *d_inputVector, *d_outputVector;
        hipMalloc((void **)&d_inputMatrix, N * N * sizeof(float));
        hipMalloc((void **)&d_inputVector, N * sizeof(float));
        hipMalloc((void **)&d_outputVector, N * sizeof(float));

        // Copy data from host to device
        hipMemcpy(d_inputMatrix, inputMatrix, N * N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_inputVector, inputVector, N * sizeof(float), hipMemcpyHostToDevice);

        // Define kernel launch configuration
        int threadsPerBlock = threadsnum;
        int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

        // Create CUDA events for timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Record start event for parallel execution time
        hipEventRecord(start);

        // Launch kernel for matrix-vector multiplication
        matrixVectorMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_outputVector, d_inputMatrix, d_inputVector, N);

        // Record stop event for parallel execution time
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        // Calculate elapsed time for parallel execution
        float parallel_milliseconds = 0;
        hipEventElapsedTime(&parallel_milliseconds, start, stop);

        // Calculate speedup
        float speedup = 2.88666f / parallel_milliseconds;

        // Print speedup for the current i
        printf("Threads: %d, Speedup: %.5f\n", threadsnum, speedup);

        // Free device memory
        hipFree(d_inputMatrix);
        hipFree(d_inputVector);
        hipFree(d_outputVector);

        // Free host memory
        free(inputMatrix);
        free(inputVector);
        free(outputVector);

        // Destroy CUDA events
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    return 0;
}
